
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <vector>

// Update bucket in parallel.
// Each thread represents a key and will increment the corresponding bucket.
__global__ void putBucket(int *key, int *bucket, int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i >= n) return;
  atomicAdd(&bucket[key[i]], 1);
}

// Prefix sum for starting indices.
// The starting index is the sum of the number of elements in all
// the buckets with a smaller index. The ending index is the starting index
// of the next bucket.
__global__ void setIndex(int *bucket, int *starting_index, int *ending_index, int *b, int range) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i >= range) return;
  for(int j=1; j<range; j<<=1) {
    b[i] = bucket[i] + starting_index[i];
    __syncthreads();
    starting_index[i] += b[i-j];
    ending_index[i-1] = starting_index[i];
    __syncthreads();
  }
}

// Change key value to the corresponding bucket id.
// Each thread represents a key id and checks in which bucket it belongs.
// Since the indices for the keys are non-overlapping,
// we can assign the values in parallel.
__global__ void setKey(int *key, int *starting_index, int *ending_index, int n, int range) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i >= n) return;
  for(int j = 0; j < range; j++) {
    if (starting_index[j] <= i && ending_index[j] > i) {
      key[i] = j;
      return;
    }
  }
}

int main() {
  // M is the number of threads per block.
  const int M = 1024;
  int n = 50;
  int range = 5;
  // Share the key array with the GPU.
  int *key;
  hipMallocManaged(&key, n*sizeof(int));
  for (int i=0; i<n; i++) {
    key[i] = rand() % range;
    printf("%d ",key[i]);
  }
  printf("\n");

  // Arrays that need to be shared with GPU.
  int *bucket;
  int *starting_index;
  int *ending_index;
  int *b;

  // Initialize arrays
  hipMallocManaged(&bucket, range*sizeof(int));
  hipMallocManaged(&starting_index, range*sizeof(int));
  hipMallocManaged(&ending_index, range*sizeof(int));
  hipMallocManaged(&b, range*sizeof(int));

  // Perform GPU computations.
  // Use all the threads in the minimum number of blocks needed.
  // This allows us to use the code for larger n and/or range.
  putBucket<<<(n+M-1)/M,M>>>(key, bucket, n);
  hipDeviceSynchronize();
  setIndex<<<(range+M-1)/M,M>>>(bucket, starting_index, ending_index, b, range);
  hipDeviceSynchronize();
  ending_index[range-1] = n;
  setKey<<<(n+M-1)/M,M>>>(key, starting_index, ending_index, n, range);
  hipDeviceSynchronize();

  // Free the space allocated to the arrays.
  hipFree(key);
  hipFree(bucket);
  hipFree(starting_index);
  hipFree(ending_index);

  for (int i=0; i<n; i++) {
    printf("%d ",key[i]);
  }
  printf("\n");
}
